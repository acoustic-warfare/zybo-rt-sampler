#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "filter_coefficients.h"


#define NX 256
#define BATCH 256
#define RANK 1

hipfftHandle plan2;
hipfftHandle plan3;
float *cuda_input; 
hipfftComplex *inputFFT;
hipfftComplex *conv;
hipfftComplex *filter_FFT;
float *res;

__global__ void spectral_convolution(hipfftComplex *filter, hipfftComplex *input, hipfftComplex *out, size_t size){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    hipfftComplex h = filter[tid];
    hipfftComplex x = input[tid];

    out[tid].x = (x.x * h.x) - (x.y * h.y);
    out[tid].y = (x.x * h.y) + (x.y * h.x);
}

__global__ void normalize(float *signal){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    signal[tid] = signal[tid] / (256);
}

void cuda_filter_init(){
    size_t filter_size_bytes = NX * sizeof(float) * 256;
    size_t filter_fft_size_bytes = 129 *sizeof(hipfftComplex) * BATCH;
    
    float *cuda_filter_coefficients;
    
    hipMalloc((void **) &cuda_filter_coefficients, filter_size_bytes);
    hipMalloc((void **) &filter_FFT, filter_fft_size_bytes);
    float *filter_coeffs = H;

    for(int i = 0; i < BATCH; i ++){
        hipMemcpy((void *) &cuda_filter_coefficients[i*BATCH], (void *) filter_coeffs, 256 * sizeof(float) , hipMemcpyHostToDevice);
    }
    
    hipfftHandle plan_filter_fft;
    hipfftResult status = hipfftCreate(&plan_filter_fft);
    if(status != 0){
        printf("CUDA_ERROR %d", status);
    }

    status = hipfftPlan1d(&plan_filter_fft, NX, HIPFFT_R2C, BATCH);
    if(status != 0){
        printf("CUDA_ERROR %d", status);
    }

    status = hipfftExecR2C(plan_filter_fft, cuda_filter_coefficients, filter_FFT);
    if(status != 0){
        printf("CUDA_ERROR %d", status);
    }
    
    

    hipFree(cuda_filter_coefficients);
    hipfftDestroy(plan_filter_fft);
}

void plan_init(){
    hipfftResult status = hipfftCreate(&plan3);
    if(status != 0){
        printf("CUDA_ERROR_1 %d", status);
    }
    status = hipfftPlan1d(&plan3, NX, HIPFFT_C2R, BATCH);
    if(status != 0){
        printf("CUDA_ERROR_2 %d", status);
    }

    status = hipfftCreate(&plan2);
    if(status != 0){
        printf("CUDA_ERROR_3 %d", status);
    }

    status = hipfftPlan1d(&plan2, NX, HIPFFT_R2C, BATCH);
    if(status != 0){
        printf("CUDA_ERROR_4 %d", status);
    }
}

void memory_init(){
    size_t input_size_bytes = NX * sizeof(float) *BATCH;
    size_t output_size_bytes = 129 * sizeof(hipfftComplex) * BATCH;
    //float *c_input;
    hipMalloc((void **) &cuda_input, input_size_bytes);
    hipMalloc((void **) &inputFFT, output_size_bytes);
    hipMalloc((void **) &conv, output_size_bytes);
    hipMalloc((void **) &res, input_size_bytes);
}

void cuda_init_all(){
    cuda_filter_init();
    plan_init();
    memory_init();
}

void cuda_filter_FFT(float *input, float *filter_output){
    size_t input_size_bytes = NX * sizeof(float) *BATCH;
    //Init output memory
    hipMemcpy((void *) cuda_input, (void *) input, input_size_bytes, hipMemcpyHostToDevice);
    
    //Perform FFT of microphone data
    hipfftResult status = hipfftExecR2C(plan2, cuda_input, inputFFT);
    if(status != 0){
        printf("CUDA_ERROR_5 %d", status);
    }
    //Perform convolution
    spectral_convolution<<<129,256>>>(filter_FFT, inputFFT, conv, 129);
    
    //Inverse FFT
    status = hipfftExecC2R(plan3, conv, res);
    if(status != 0){
        printf("CUDA_ERROR_6 %d", status);
    }
    normalize<<<256, 256>>>(res);
    hipMemcpy(filter_output, res, input_size_bytes, hipMemcpyDeviceToHost);
}